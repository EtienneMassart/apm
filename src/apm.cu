/**
 * APPROXIMATE PATTERN MATCHING
 *
 * INF560
 */
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <unistd.h>
#include <omp.h>

#include <hip/hip_runtime.h>

#define APM_DEBUG 0

void cas0_OpenMP(int nb_patterns, char ** pattern, int n_bytes, int approx_factor, char * buf, int * n_matches);
void cas1_OpenMP(int nb_patterns, char ** pattern, int n_bytes, int approx_factor, char * buf, int * n_matches);
void cas2_OpenMP(int nb_patterns, char ** pattern, int n_bytes, int approx_factor, char * buf, int * n_matches);
void cas1_Cuda(int nb_patterns, char ** pattern, int n_bytes, int approx_factor, char * buf, int * n_matches);

char *read_input_file(char *filename, int *size) {
    char *buf;
    off_t fsize;
    int fd = 0;
    int n_bytes = 1;

    /* Open the text file */
    fd = open(filename, O_RDONLY);
    if (fd == -1) {
        fprintf(stderr, "Unable to open the text file <%s>\n", filename);
        return NULL;
    }

    /* Get the number of characters in the textfile */
    fsize = lseek(fd, 0, SEEK_END);
    if (fsize == -1) {
        fprintf(stderr, "Unable to lseek to the end\n");
        return NULL;
    }

#if APM_DEBUG
    printf("File length: %lld\n", fsize);
#endif

    /* Go back to the beginning of the input file */
    if (lseek(fd, 0, SEEK_SET) == -1) {
        fprintf(stderr, "Unable to lseek to start\n");
        return NULL;
    }

    /* Allocate data to copy the target text */
    buf = (char *)malloc(fsize * sizeof(char));
    if (buf == NULL) {
        fprintf(stderr, "Unable to allocate %ld byte(s) for main array\n",
                fsize);
        return NULL;
    }

    n_bytes = read(fd, buf, fsize);
    if (n_bytes != fsize) {
        fprintf(
            stderr,
            "Unable to copy %ld byte(s) from text file (%d byte(s) copied)\n",
            fsize, n_bytes);
        return NULL;
    }

#if APM_DEBUG
    printf("Number of read bytes: %d\n", n_bytes);
#endif

    *size = n_bytes;

    close(fd);

    return buf;
}

#define MIN3(a, b, c) \
    ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)))

__host__ __device__ int levenshtein(char *s1, char *s2, int len, int *column) {
    unsigned int x, y, lastdiag, olddiag;

    for (y = 1; y <= len; y++) {
        column[y] = y;
    }
    for (x = 1; x <= len; x++) {
        column[0] = x;
        lastdiag = x - 1;
        for (y = 1; y <= len; y++) {
            olddiag = column[y];
            column[y] = MIN3(column[y] + 1, column[y - 1] + 1,
                             lastdiag + (s1[y - 1] == s2[x - 1] ? 0 : 1));
            lastdiag = olddiag;
        }
    }
    return (column[len]);
}

int main(int argc, char **argv) {
    char **pattern;
    char *filename;
    int approx_factor = 0;
    int nb_patterns = 0;
    int i ;
    char *buf;
    struct timeval t1, t2;
    double duration;
    int n_bytes;
    int *n_matches;

    /* Check number of arguments */
    if (argc < 4) {
        printf(
            "Usage: %s approximation_factor "
            "dna_database pattern1 pattern2 ...\n",
            argv[0]);
        return 1;
    }

    /* Get the distance factor */
    approx_factor = atoi(argv[1]);

    /* Grab the filename containing the target text */
    filename = argv[2];

    /* Get the number of patterns that the user wants to search for */
    nb_patterns = argc - 3;

    /* Fill the pattern array */
    pattern = (char **)malloc(nb_patterns * sizeof(char *));
    if (pattern == NULL) {
        fprintf(stderr, "Unable to allocate array of pattern of size %d\n",
                nb_patterns);
        return 1;
    }

    /* Grab the patterns */
    for (i = 0; i < nb_patterns; i++) {
        int l;

        l = strlen(argv[i + 3]);
        if (l <= 0) {
            fprintf(stderr, "Error while parsing argument %d\n", i + 3);
            return 1;
        }

        pattern[i] = (char *)malloc((l + 1) * sizeof(char));
        if (pattern[i] == NULL) {
            fprintf(stderr, "Unable to allocate string of size %d\n", l);
            return 1;
        }

        strncpy(pattern[i], argv[i + 3], (l + 1));
    }

    printf(
        "Approximate Pattern Mathing: "
        "looking for %d pattern(s) in file %s w/ distance of %d\n",
        nb_patterns, filename, approx_factor);

    buf = read_input_file(filename, &n_bytes);
    if (buf == NULL) {
        return 1;
    }

    /* Allocate the array of matches */
    n_matches = (int *)malloc(nb_patterns * sizeof(int));
    if (n_matches == NULL) {
        fprintf(stderr, "Error: unable to allocate memory for %ldB\n",
                nb_patterns * sizeof(int));
        return 1;
    }

    /*****
     * BEGIN MAIN LOOP
     ******/

    /* Timer start */
    gettimeofday(&t1, NULL);

    /* Check each pattern one by one */
    // cas1_OpenMP(nb_patterns, pattern, n_bytes, approx_factor, buf, n_matches);
    cas1_Cuda(nb_patterns, pattern, n_bytes, approx_factor, buf, n_matches);


    /* Timer stop */
    gettimeofday(&t2, NULL);

    duration = (t2.tv_sec - t1.tv_sec) + ((t2.tv_usec - t1.tv_usec) / 1e6);

    printf("APM done in %lf s\n", duration);

    /*****
     * END MAIN LOOP
     ******/

    for (i = 0; i < nb_patterns; i++) {
        printf("Number of matches for pattern <%s>: %d\n", pattern[i],
               n_matches[i]);
    }

    return 0;
}

void cas0_OpenMP(int nb_patterns, char ** pattern, int n_bytes, int approx_factor, char * buf, int * n_matches){
    for (int i = 0; i < nb_patterns; i++) {
        // printf("Processing with OpenMP thread %d\n", omp_get_thread_num());
        int size_pattern = strlen(pattern[i]);
        int *column;

        /* Initialize the number of matches to 0 */
        n_matches[i] = 0;

        column = (int *)malloc((size_pattern + 1) * sizeof(int));
        // if (column == NULL) {
        //     fprintf(stderr,
        //             "Error: unable to allocate memory for column (%ldB)\n",
        //             (size_pattern + 1) * sizeof(int));
        //     return 1;
        // }

        /* Traverse the input data up to the end of the file */
        for (int j = 0; j < n_bytes; j++) {
            int distance = 0;
            int size;

#if APM_DEBUG
            if (j % 100 == 0) {
                printf("Procesing byte %d (out of %d)\n", j, n_bytes);
            }
#endif

            size = size_pattern;
            if (n_bytes - j < size_pattern) {
                size = n_bytes - j;
            }

            distance = levenshtein(pattern[i], &buf[j], size, column);

            if (distance <= approx_factor) {
                n_matches[i]++;
            }
        }

        free(column);
    }
}

void cas1_OpenMP(int nb_patterns, char ** pattern, int n_bytes, int approx_factor, char * buf, int * n_matches){
    for (int i = 0; i < nb_patterns; i++) {
        // printf("Processing with OpenMP thread %d\n", omp_get_thread_num());
        int size_pattern = strlen(pattern[i]);
        int *column;

        /* Initialize the number of matches to 0 */
        n_matches[i] = 0;

        // if (column == NULL) {
        //     fprintf(stderr,
        //             "Error: unable to allocate memory for column (%ldB)\n",
        //             (size_pattern + 1) * sizeof(int));
        //     return 1;
        // }

        /* Traverse the input data up to the end of the file */
# pragma omp parallel for private(column) reduction(+:n_matches[i])
        for (int j = 0; j < n_bytes; j++) {
            // on est obligé d'allouer l'espace pour la column plus tard pour la rendre privée
            column = (int *)malloc((size_pattern + 1) * sizeof(int));
            int distance = 0;
            int size;

#if APM_DEBUG
            if (j % 100 == 0) {
                printf("Procesing byte %d (out of %d)\n", j, n_bytes);
            }
#endif

            size = size_pattern;
            if (n_bytes - j < size_pattern) {
                size = n_bytes - j;
            }

            distance = levenshtein(pattern[i], &buf[j], size, column);

            if (distance <= approx_factor) {
                n_matches[i]++;
            }
            free(column);
        }

    }
}


void cas2_OpenMP(int nb_patterns, char ** pattern, int n_bytes, int approx_factor, char * buf, int * n_matches){
    #pragma omp parallel for
    for (int i = 0; i < nb_patterns; i++) {
        // printf("Processing with OpenMP thread %d\n", omp_get_thread_num());
        int size_pattern = strlen(pattern[i]);
        int *column;

        /* Initialize the number of matches to 0 */
        n_matches[i] = 0;

        column = (int *)malloc((size_pattern + 1) * sizeof(int));
        // if (column == NULL) {
        //     fprintf(stderr,
        //             "Error: unable to allocate memory for column (%ldB)\n",
        //             (size_pattern + 1) * sizeof(int));
        //     return 1;
        // }

        /* Traverse the input data up to the end of the file */
        for (int j = 0; j < n_bytes; j++) {
            int distance = 0;
            int size;

#if APM_DEBUG
            if (j % 100 == 0) {
                printf("Procesing byte %d (out of %d)\n", j, n_bytes);
            }
#endif

            size = size_pattern;
            if (n_bytes - j < size_pattern) {
                size = n_bytes - j;
            }

            distance = levenshtein(pattern[i], &buf[j], size, column);

            if (distance <= approx_factor) {
                n_matches[i]++;
            }
        }

        free(column);
    }
}


__global__ void kernel_calcul(int n_bytes, int size_pattern, int approx_factor, char * buf, char * pattern, int * distances){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n_bytes){
        int size;

        size = size_pattern;
        if (n_bytes - i < size_pattern) {
            size = n_bytes - i;
        }
        int * column = (int *)malloc((size_pattern + 1) * sizeof(int));

        distances[i] = levenshtein(pattern, &buf[i], size, column);
        free(column);
    }
}


void cas1_Cuda(int nb_patterns, char ** pattern, int n_bytes, int approx_factor, char * buf, int * n_matches){
    for (int i = 0; i < nb_patterns; i++) {
        // printf("Processing with OpenMP thread %d\n", omp_get_thread_num());
        int size_pattern = strlen(pattern[i]);

        /* Initialize the number of matches to 0 */
        n_matches[i] = 0;

        /* Traverse the input data up to the end of the file */
        int * distances = (int *)malloc(n_bytes * sizeof(int));
        char * d_buf;
        char * d_pattern;
        int * d_distances;

        hipMalloc((void **) &d_buf, n_bytes * sizeof(char));
        hipMalloc((void **) &d_pattern, size_pattern * sizeof(char));
        hipMalloc((void **) &d_distances, n_bytes * sizeof(int));
        hipMemcpy(d_buf, buf, n_bytes * sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(d_pattern, pattern[i], size_pattern * sizeof(char), hipMemcpyHostToDevice);

        dim3 Db = dim3(256, 1, 1);
        dim3 Dg = dim3((n_bytes + Db.x - 1) / Db.x, 1, 1);

        kernel_calcul<<<Dg,Db>>>(n_bytes, size_pattern, approx_factor, d_buf, d_pattern, d_distances);

        hipMemcpy(distances, d_distances, n_bytes * sizeof(int), hipMemcpyDeviceToHost);

        for (int j = 0; j < n_bytes; j++) {
            if (distances[j] <= approx_factor) {
                n_matches[i]++;
            }
        }
    }
}

